#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "utils.h"
#include "cuda_utils.h"
#include <vector>
#include <random>
#include <iostream>
#include <chrono>
#include <string>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

using namespace std;


__global__ void precalc_interp_kick(const float * voltage_array,
                                    const float * bin_centers,
                                    float * volt_kick,
                                    float * factor,
                                    const int bins,
                                    const float acc_kick)
{
    const float inv_bin_width = (bins - 1) / (bin_centers[bins - 1] - bin_centers[0]);

    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x;
            i < bins - 1;
            i += blockDim.x * gridDim.x) {
        volt_kick[i] = (voltage_array[i + 1] - voltage_array[i]) * inv_bin_width;
        factor[i] = voltage_array[i] - bin_centers[i] * volt_kick[i] + acc_kick;
    }
}


__global__ void linear_interp_kick(const float * input,
                                   float * output,
                                   const float * volt_kick,
                                   const float * factor,
                                   const float * bin_centers,
                                   const int bins,
                                   const int n,
                                   const float acc_kick)
{
    const float center0 = bin_centers[0];
    const float inv_bin_width = (bins - 1) / (bin_centers[bins - 1] - center0);

    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x;
            i < n;
            i += blockDim.x * gridDim.x) {
        unsigned bin = (unsigned) floor((input[i] - center0) * inv_bin_width);
        if (bin < bins - 1)
            output[i] += input[i] * volt_kick[bin] + factor[bin];
        else
            output[i] += acc_kick;

    }

}


int main(int argc, char const *argv[])
{
    int n_turns = 50000;
    int n_particles = 1000000;
    int n_slices = 1000;
    int blocks = 512;
    int threads = 1024;

    if (argc > 1) n_turns = atoi(argv[1]);
    if (argc > 2) n_particles = atoi(argv[2]);
    if (argc > 3) n_slices = atoi(argv[3]);
    if (argc > 4) blocks = atoi(argv[4]);
    if (argc > 5) threads = atoi(argv[5]);

    // setup random engine
    default_random_engine gen;
    uniform_real_distribution<float> d(0.0, 1.0);

    // initialize variables
    vector<float> dE, dt;
    vector<float> voltage, edges, bin_centers;
    float cut_left, cut_right, acc_kick;

    string input = HOME "/input_files/distribution_10M_particles.txt";
    read_distribution(input, n_particles, dt, dE);

    voltage.resize(n_slices);
    for (int i = 0; i < n_slices; ++i) {
        voltage[i] = d(gen);
    }
    cut_left = 1.05 * (*min_element(dt.begin(), dt.end()));
    cut_right = 0.95 * (*max_element(dt.begin(), dt.end()));

    // cut_left = dt[rand() % n_slices];
    // cut_right = dt[rand() % n_slices];
    acc_kick = 10e6 * d(gen);
    if (cut_left > cut_right) swap(cut_left, cut_right);

    edges.resize(n_slices);
    linspace(cut_left, cut_right, n_slices + 1, edges.data());

    bin_centers.resize(n_slices);
    for (int i = 0; i < n_slices; ++i) {
        bin_centers[i] = (edges[i] + edges[i + 1]) / 2.;
    }

    thrust::device_vector<float> d_dE = dE;
    thrust::device_vector<float> d_dt = dt;
    thrust::device_vector<float> d_voltage = voltage;
    thrust::device_vector<float> d_bin_centers = bin_centers;
    thrust::device_vector<float> d_volt_kick(n_slices - 1);
    thrust::device_vector<float> d_factor(n_slices - 1);

    float *d_dE_ptr = thrust::raw_pointer_cast(d_dE.data());
    float *d_dt_ptr = thrust::raw_pointer_cast(d_dt.data());
    float *d_bin_centers_ptr = thrust::raw_pointer_cast(d_bin_centers.data());
    float *d_voltage_ptr = thrust::raw_pointer_cast(d_voltage.data());
    float *d_volt_kick_ptr = thrust::raw_pointer_cast(d_volt_kick.data());
    float *d_factor_ptr = thrust::raw_pointer_cast(d_factor.data());

    auto start = chrono::high_resolution_clock::now();
    // main loop
    for (int i = 0; i < n_turns; ++i) {
        precalc_interp_kick <<< (n_slices + 63) / 64, 64 >>> (
            d_voltage_ptr,
            d_bin_centers_ptr,
            d_volt_kick_ptr,
            d_factor_ptr,
            n_slices, acc_kick);
        linear_interp_kick <<< blocks, threads>>> (d_dt_ptr,
                d_dE_ptr,
                d_volt_kick_ptr,
                d_factor_ptr,
                d_bin_centers_ptr,
                n_slices, n_particles, acc_kick);
        hipDeviceSynchronize();
    }

    auto end = chrono::high_resolution_clock::now();
    thrust::copy(d_dE.begin(), d_dE.end(), dE.begin());
    auto duration = chrono::duration_cast<chrono::milliseconds>(end - start).count();
    printf("function\tcounter\taverage_value\tstd(%%)\tcalls\n");
    printf("interp_kick_gpu_v10\ttime(ms)\t%d\t0\t1\n", duration);
    printf("dE: %lf\n", accumulate(dE.begin(), dE.end(), 0.0) / n_particles);
    // papiprof->stop_counters();
    // papiprof->report_timing();
    // report results

    return 0;
}